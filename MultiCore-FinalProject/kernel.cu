#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <vector>
#include <Windows.h>
#include <stdlib.h>
#include <unordered_map>
#include <assert.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <omp.h>
#include <string>

using namespace std;

struct Matrix {
	double* A;
	int m;
};

vector<string> get_all_files_names_within_folder(string folder_name)
{
	vector<string> names;
	string search_path = folder_name + "/*.txt*";
	WIN32_FIND_DATA fd;
	HANDLE hFind = ::FindFirstFile(search_path.c_str(), &fd);
	if (hFind != INVALID_HANDLE_VALUE) {
		do {
			// read all (real) files in current folder
			// , delete '!' read other 2 default folder . and ..
			if (!(fd.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY)) {
				//auto a = fd.cFileName;
				names.push_back(folder_name + "\\" + fd.cFileName);
			}
		} while (::FindNextFile(hFind, &fd));
		::FindClose(hFind);
	}
	return names;
}

void printMatrix(int m, int n, const double* A, int lda, const char* name)
{
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			double Areg = A[row + col * lda];
			cout << Areg << " ";
			//printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
		}
		cout << endl;
	}
}

double call_to_cusolver_with_stream(Matrix matrix, hipStream_t& stream) {

	hipsolverHandle_t cusolverH = NULL;
	//hipStream_t stream = NULL;

	hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	hipError_t cudaStat4 = hipSuccess;
	int m = matrix.m;
	int lda = m;

	double* A = matrix.A;
	double* LU = new double[lda * m];
	int* Ipiv = new int[m];      /* host copy of pivoting sequence */
	int info = 0;     /* host copy of error info */

	double* d_A = NULL; /* device copy of A */
	int* d_info = NULL; /* error info */
	int  lwork = 0;     /* size of workspace */
	double* d_work = NULL; /* device workspace for getrf */

	printf("pivot is off: compute A = L*U (not numerically stable)\n");

	printf("A = \n");
	printMatrix(m, m, A, lda, "A");
	printf("=====\n");

	/* step 1: create cusolver handle, bind a stream */
	status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	//cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	//assert(hipSuccess == cudaStat1);

	status = hipsolverSetStream(cusolverH, stream);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	/* step 2: copy A to device */
	cudaStat1 = hipMalloc((void**)&d_A, sizeof(double) * lda * m);
	cudaStat4 = hipMalloc((void**)&d_info, sizeof(int));
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	assert(hipSuccess == cudaStat4);

	cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);

	/* step 3: query working space of getrf */
	status = hipsolverDnDgetrf_bufferSize(
		cusolverH,
		m,
		m,
		d_A,
		lda,
		&lwork);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	cudaStat1 = hipMalloc((void**)&d_work, sizeof(double) * lwork);
	assert(hipSuccess == cudaStat1);

	/* step 4: LU factorization */

	status = hipsolverDnDgetrf(
		cusolverH,
		m,
		m,
		d_A,
		lda,
		d_work,
		NULL,
		d_info);
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	assert(hipSuccess == cudaStat1);

	cudaStat2 = hipMemcpy(LU, d_A, sizeof(double) * lda * m, hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	if (0 > info) {
		printf("%d-th parameter is wrong \n", -info);
		exit(1);
	}
	printf("L and U = (matlab base-1)\n");
	printMatrix(m, m, LU, lda, "LU");
	printf("=====\n");

	double determinant = 1.0;
#pragma omp parallel for reduction(*: determinant)
	for (int i = 0; i < lda * lda; i += lda) {
		determinant *= LU[i];
	}

	/* free resources */
	if (d_A) hipFree(d_A);
	if (d_info) hipFree(d_info);
	if (d_work) hipFree(d_work);

	if (cusolverH) hipsolverDnDestroy(cusolverH);
	return determinant;
}

Matrix string_to_matrix(string matrix_str) {
	int len = matrix_str.length();
	// Assuming no \0 at its end.
	int n = (len - 1) / 2;
	double* A = new double[n];
	for (int i = 0; i < len; i += 2) {
		A[i / 2] = matrix_str[i] - '0';
		cout << A[i / 2] << " ";
	}
	Matrix matrix = {
		A,
		sqrt(n)
	};
	return matrix;
}

unordered_map<string, vector<double>> call_cuda(unordered_map<string, vector<Matrix>> matrices_of_files) {
	unordered_map<string, vector<double>> file_determinants;
	for (auto const& x : matrices_of_files) {
		hipStream_t stream;
		hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
		vector<double> dets_of_file;
#pragma omp parallel for
		for (int i = 0; i < x.second.size(); i++) {
			Matrix matrix = x.second.at(i);
			double det = call_to_cusolver_with_stream(matrix, stream);
#pragma omp critical
			dets_of_file.push_back(det);
		}
		if (stream) hipStreamDestroy(stream);
		file_determinants[x.first] = dets_of_file;
	}
	return file_determinants;
}

int main(int argc, char* argv[])
{
	vector<string> files = get_all_files_names_within_folder("data_in");
	unordered_map<string, vector<Matrix>> matrices_of_files;
#pragma omp parallel for
	for (int i = 0; i < files.size(); i++) {
		string file_name = files.at(i);
		vector<Matrix> file_matrices;
		ifstream input(file_name);
		for (string line; getline(input, line);) {
			if (line.length() > 2) {
				file_matrices.push_back(string_to_matrix(line));
			}
		}
//#pragma omp critical
		matrices_of_files[file_name] = file_matrices;
	}


	//call_to_cusolver_with_stream(4);
	//hipDeviceReset();
	return 0;
}