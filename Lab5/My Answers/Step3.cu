#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

hipError_t addWithCuda_1(int* c, const int* a, const int* b, unsigned int matSizeX, unsigned int matSizeY, unsigned int N, int baseSize);
hipError_t addWithCuda_2(int* c, const int* a, const int* b, unsigned int matSizeX, unsigned int matSizeY, unsigned int N, int baseSize);
void fillMat(int* v, int matSizeX, int matSizeY);
void printMat(int* v, int matSizeX, int matSizeY);


__global__ void addKernel_nSums(int* c, const int* a, const int* b, unsigned int N)
{

	int threadID = threadIdx.x + (threadIdx.y * blockDim.x);
	threadID *= N;

	for (int x = threadID; x < N + threadID; x++)
	{
		c[x] = a[x] + b[x];
	}
}


__global__ void addKernel_nBlocks(int* c, const int* a, const int* b)
{
	const int blockId = blockIdx.x //1D
		+ blockIdx.y * gridDim.x //2D
		+ gridDim.x * gridDim.y * blockIdx.z; //3D
	const int threadId = threadIdx.x //1D
		+ threadIdx.y * blockDim.x //2D
		+ blockDim.x * blockDim.y * threadIdx.z; //3D
	const int globalThreadID = blockId * blockDim.x * blockDim.y * blockDim.z + threadId;

	c[globalThreadID] = a[globalThreadID] + b[globalThreadID];
}


int main()
{
	const int squareOfN = 10;
	const int baseSize = 32;
	const int matSizeX = baseSize * squareOfN;
	const int matSizeY = baseSize * squareOfN;
	int* a;
	int* b;
	int* c;
	int* d;
	a = (int*)malloc(sizeof(int) * matSizeX * matSizeY);
	b = (int*)malloc(sizeof(int) * matSizeX * matSizeY);
	c = (int*)malloc(sizeof(int) * matSizeX * matSizeY);
	d = (int*)malloc(sizeof(int) * matSizeX * matSizeY);

	fillMat(a, matSizeX, matSizeY);
	fillMat(b, matSizeX, matSizeY);


	// Add vectors in parallel.
	printf("N Blocks: \n");
	hipError_t cudaStatus = addWithCuda_1(c, a, b, matSizeX, matSizeY, squareOfN * squareOfN, baseSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("N Sums: \n");
	cudaStatus = addWithCuda_2(d, a, b, matSizeX, matSizeY, squareOfN * squareOfN, baseSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	 /*printMat(a, matSizeX, matSizeY);
	 printMat(b, matSizeX, matSizeY);
	printMat(d, matSizeX, matSizeY);*/

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
void fillMat(int* v, int matSizeX, int matSizeY) {
	static int L = 0;
	for (int i = 0; i < matSizeX; i++) {
		for (int j = 0; j < matSizeY; j++)
			v[i * matSizeY + j] = L++;
	}
}
void printMat(int* v, int matSizeX, int matSizeY) {
	int i;
	printf("[-] Vector elements: \n");
	for (int i = 0; i < matSizeX; i++) {
		for (int j = 0; j < matSizeY; j++)
			printf("%d	", v[i * matSizeY + j]);
		printf("\n");

	}
	printf("\b\b  \n");
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda_1(int* c, const int* a, const int* b, unsigned int matSizeX, unsigned int matSizeY, unsigned int N, int baseSize)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;
	hipEvent_t start;
	hipEventCreate(&start);
	hipEvent_t stop;
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	hipEventRecord(start, NULL);
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, matSizeX * matSizeY * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_a, matSizeX * matSizeY * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_b, matSizeX * matSizeY * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, matSizeX * matSizeY * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, matSizeX * matSizeY * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// KEEP THIS HERE
	//dim3 block_size = dim3(matSizeX, matSizeY / N, 1);
	//// Launch a kernel on the GPU with one thread for each element.
	//addKernel << <1, block_size >> > (dev_c, dev_a, dev_b, N);

	/*dim3 block_size = dim3(matSizeX, matSizeY / baseSize, 1);
	addKernel_nSums << <1, block_size >> > (dev_c, dev_a, dev_b, N);*/

	addKernel_nBlocks << <N, 1024 >> > (dev_c, dev_b, dev_a);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, matSizeX * matSizeY * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	hipEventRecord(stop, NULL);
	cudaStatus = hipEventSynchronize(stop);
	float mSecTotal;
	cudaStatus = hipEventElapsedTime(&mSecTotal, start, stop);
	printf("Time: %f \n\n", mSecTotal);

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
hipError_t addWithCuda_2(int* c, const int* a, const int* b, unsigned int matSizeX, unsigned int matSizeY, unsigned int N, int baseSize)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;
	hipEvent_t start;
	hipEventCreate(&start);
	hipEvent_t stop;
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	hipEventRecord(start, NULL);
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, matSizeX * matSizeY * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_a, matSizeX * matSizeY * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_b, matSizeX * matSizeY * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, matSizeX * matSizeY * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, matSizeX * matSizeY * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// KEEP THIS HERE
	//dim3 block_size = dim3(matSizeX, matSizeY / N, 1);
	//// Launch a kernel on the GPU with one thread for each element.
	//addKernel << <1, block_size >> > (dev_c, dev_a, dev_b, N);

	dim3 block_size = dim3(matSizeX / baseSize, matSizeY / baseSize, 1);
	addKernel_nSums << <1, block_size >> > (dev_c, dev_a, dev_b, N);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, matSizeX * matSizeY * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	hipEventRecord(stop, NULL);
	cudaStatus = hipEventSynchronize(stop);
	float mSecTotal;
	cudaStatus = hipEventElapsedTime(&mSecTotal, start, stop);
	printf("Time: %f \n\n", mSecTotal);

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}