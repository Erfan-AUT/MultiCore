#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128 

#include <thrust/device_vector.h>
#include "hipblas.h"
#include <iostream>

using namespace std;

// C-style indexing
int ci(int row, int column, int nColumns) {
    return row * nColumns + column;
}

int main(void)
{
    size_t n = 1;
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cerr << "!!!! CUBLAS initialization error\n";
    }
    while (n > 0)
    {
        printf("[-] N = ");
        scanf("%u", &n);
        printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", n, n, n, n);

        // initialize data
        thrust::device_vector<float> D(n * n);
        thrust::device_vector<float> E(n * n);
        thrust::device_vector<float> F(n * n);
        for (size_t i = 0; i < n; i++) {
            for (size_t j = 0; j < n; j++) {
                D[ci(i, j, n)] = 1;
                //cout << D[ci(i, j, n)] << " ";
            }
            //cout << "\n";
        }

        for (size_t i = 0; i < n; i++) {
            for (size_t j = 0; j < n; j++) {
                E[ci(i, j, n)] = 0.1;
                //cout << E[ci(i, j, n)] << " ";
            }
            //cout << "\n";
        }

        for (size_t i = 0; i < n; i++)
            for (size_t j = 0; j < n; j++)
                F[ci(i, j, n)] = 0;

        /* Initialize CUBLAS */
        hipEvent_t start;
        hipError_t error = hipEventCreate(&start);
        hipEvent_t stop;
        error = hipEventCreate(&stop);
        

        float alpha = 1.0f, beta = 0.0f;
        error = hipEventRecord(start, NULL);
        status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, n, n,
            &alpha, thrust::raw_pointer_cast(&E[0]), n,
            thrust::raw_pointer_cast(&D[0]), n,
            &beta, thrust::raw_pointer_cast(&F[0]), n);// colE  x rowD
        if (status != HIPBLAS_STATUS_SUCCESS) {
            cerr << "!!!! kernel execution error.\n";
        }
        error = hipEventRecord(stop, NULL);
        error = hipEventSynchronize(stop);
        float msecTotal = 0.0f;
        error = hipEventElapsedTime(&msecTotal, start, stop);
        cout << "Elapsed time in msec = " << msecTotal << endl;

        /*for (size_t i = 0; i < n; i++) {
            for (size_t j = 0; j < n; j++) {
                cout << F[ci(i, j, n)] << " ";
            }
            cout << "\n";
        }*/

    }
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cerr << "!!!! shutdown error (A)\n";
    }

    return 0;
}