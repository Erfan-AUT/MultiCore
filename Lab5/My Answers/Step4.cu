#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t addWithCuda(int* output, int size);

__global__ void addKernel(int* output)
{
	const int blockId = blockIdx.x //1D
		+ blockIdx.y * gridDim.x //2D
		+ gridDim.x * gridDim.y * blockIdx.z; //3D
	const int threadId = threadIdx.x //1D
		+ threadIdx.y * blockDim.x //2D
		+ blockDim.x * blockDim.y * threadIdx.z; //3D
	const int warpID = threadId / warpSize;
	const int globalThreadID = blockId * blockDim.x * blockDim.y * blockDim.z + threadId;
	const int arrIdx = globalThreadID * 4;
	output[arrIdx] = globalThreadID;
	output[arrIdx + 1] = blockId;
	output[arrIdx + 2] = warpID;
	output[arrIdx + 3] = threadId;
}

int main()
{
	const int size = 128;
	const int size2 = 4;
	int* output = new int[size * size2];
	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(output, size * size2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	for (int i = 0; i < size * size2; i += 4)
	{
		printf("Calculated Thread: %d - Block: %d - Warp: %d - Thread: %d \n", output[i], output[i+1], output[i+2], output[i+3]);
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* output, int size)
{
	int* dev_out = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_out, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_out, output, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <2, 64 >> > (dev_out);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_out, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_out);

	return cudaStatus;
}

