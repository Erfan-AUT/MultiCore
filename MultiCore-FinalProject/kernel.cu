
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <vector>
#include <Windows.h>
#include <stdlib.h>
#include <unordered_map>
#include <assert.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <omp.h>
#include <string>

using namespace std;

vector<string> get_all_files_names_within_folder(string folder)
{
    vector<string> names;
    string search_path = folder + "/*.txt*";
    WIN32_FIND_DATA fd;
    HANDLE hFind = ::FindFirstFile(search_path.c_str(), &fd);
    if (hFind != INVALID_HANDLE_VALUE) {
        do {
            // read all (real) files in current folder
            // , delete '!' read other 2 default folder . and ..
            if (!(fd.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY)) {
                names.push_back(fd.cFileName);
            }
        } while (::FindNextFile(hFind, &fd));
        ::FindClose(hFind);
    }
    return names;
}


void printMatrix(int m, int n, const double* A, int lda, const char* name)
{
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            double Areg = A[row + col * lda];
            cout << Areg << " ";
            //printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
        }
        cout << endl;
    }
}

void call_to_cusolver_with_stream(int m) {

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    int lda = m;

    double* A = new double[lda * m] { 1.0, 4.0, 7.0, 2.0, 5.0, 8.0, 3.0, 6.0, 10.0, 1.0, 4.0, 7.0, 2.0, 5.0, 8.0, 4.0 };
    double* LU = new double[lda * m];
    int* Ipiv = new int[m];      /* host copy of pivoting sequence */
    int info = 0;     /* host copy of error info */

    double* d_A = NULL; /* device copy of A */
    int* d_info = NULL; /* error info */
    int  lwork = 0;     /* size of workspace */
    double* d_work = NULL; /* device workspace for getrf */

    printf("pivot is off: compute A = L*U (not numerically stable)\n");

    printf("A = \n");
    printMatrix(m, m, A, lda, "A");
    printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    /* step 2: copy A to device */
    cudaStat1 = hipMalloc((void**)&d_A, sizeof(double) * lda * m);
    cudaStat4 = hipMalloc((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    /* step 3: query working space of getrf */
    status = hipsolverDnDgetrf_bufferSize(
        cusolverH,
        m,
        m,
        d_A,
        lda,
        &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double) * lwork);
    assert(hipSuccess == cudaStat1);

    /* step 4: LU factorization */

    status = hipsolverDnDgetrf(
        cusolverH,
        m,
        m,
        d_A,
        lda,
        d_work,
        NULL,
        d_info);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat2 = hipMemcpy(LU, d_A, sizeof(double) * lda * m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    if (0 > info) {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    printf("L and U = (matlab base-1)\n");
    printMatrix(m, m, LU, lda, "LU");
    printf("=====\n");


    /* free resources */
    if (d_A) hipFree(d_A);
    if (d_info) hipFree(d_info);
    if (d_work) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream) hipStreamDestroy(stream);

}



int main(int argc, char* argv[])
{
    vector<string> files = get_all_files_names_within_folder("in_all");
    unordered_map<string, vector<string>> strings_of_files;
#pragma omp parallel for
    for (string file_name : files) {
        vector<string> file_strings;
        ifstream input(file_name);
        for (string line; getline(input, line)) {
            file_strings.push_back(line);
        }
#pragma omp critical
        strings_of_files[file_name] = file_strings;
    }
    call_to_cusolver_with_stream(4);
    hipDeviceReset();
    return 0;
}